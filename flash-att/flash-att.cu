#include "hip/hip_runtime.h"
#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "./common.h"
#include <stdint.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <math.h>

#define DOUBLE(pointer) (reinterpret_cast<double*>(&(pointer))[0])
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define float16_t half
//#define printf(...);

void Mimo64_alloc_host_mem(void** host_ptr_addr, size_t size)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    err = hipHostMalloc((void **)host_ptr_addr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate host memory (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	return;
}

void Mimo64_free_host_mem(void* host_ptr)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    if (host_ptr != NULL)
    {
        err = hipHostFree(host_ptr);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to free host memory (error code %s)!\n",
                    hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
	
	return;
}

void Mimo64_alloc_device_mem(void** dev_ptr_addr, size_t size)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
  
	err = hipMalloc((void**)dev_ptr_addr, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate cuda device mem (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	return;
}

void Mimo64_free_device_mem(void* dev_ptr)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    if (dev_ptr != NULL){
		err = hipFree(dev_ptr);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to free cuda device mem (error code %s)!\n",
                    hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	}

	return;
}


void initialData_f32(float *ip, int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)rand() / (float)RAND_MAX;
		//printf("val[%d]: %d \n", i, ip[i]);
    }

	return;
}


void initialData_Y_f32(float *ip, int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
		if (i < size)
			ip[i] = (float) 1.0f; //(rand() & 0xFF);
		//printf("val[%d]: %d \n", i, ip[i]);
		else
			ip[i] = (float)0.0; //(rand() & 0xFF);
		//printf("val[%d]: %d \n", i, ip[i]);
    }

	return;
}

void Mimo64_init_device_const_mem()
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

 

	return;
}


void float2half(float16_t *dst, float *src, int nElem){
	for (int i = 0; i < nElem; i++){
		dst[i] = __float2half(src[i]);
	}
	
	return;
}

hipStream_t *streams;

void Mimo64_createStreams(int numOfStreams){

	streams = (hipStream_t *)malloc(numOfStreams * sizeof(hipStream_t));

	for (int i = 0; i < numOfStreams; i++)
		hipStreamCreate(&streams[i]);

	return;
}


void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-2;
    bool match = 1;
	
//	printf("hello world. \n");
//	printf("host: %f gpu: %f \n", hostRef[0], gpuRef[0]);
    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %f gpu %f at %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");
}


void flash_att_naive_kernel(float* Q,
							float* K,
							float* V,
							float* O,
							float* S,
							int nBatch,
							int nHead,
							int N,
							int d) 
{
	int iBatch, iHead, iN, id;
	int iRow, jCol;
	
	for (iBatch = 0; iBatch < nBatch; iBatch++){
		for (iHead = 0; iHead < nHead; iHead++){
			// for each batch, length = nHead * N * d
			// for each head, length = N * d
			int Offset = iBatch * nHead * N * d + iHead * N * d;
			float* Q_1 = Q + Offset;
			float* K_1 = K + Offset;
			float* V_1 = V + Offset;
			float* O_1 = O + Offset;
			float* S_1 = S; // + Offset_QK_T;
			
			// Step 1: Q * K_T
			for (iRow = 0; iRow < N; iRow++){
				for (jCol = 0; jCol < N; jCol++){
					float* Q_2 = Q_1 + iRow * d;
					float* K_2 = K_1 + jCol * d;
					float* S_2 = S_1 + iRow * N + jCol;

					float accu = 0.0f;
					for (id = 0; id < d; id++)
					{
						float Q_val = *(Q_2 + id);
						float K_val = *(K_2 + id);
						accu += Q_val * K_val;
					}

					// printf("iRow: %d, jCol: %d, S: %f \n", iRow, jCol, accu);
					*S_2 = accu;
				}
			}
			
			
			// Step 2: Softmax
			for (iRow = 0; iRow < N; iRow++){
				float* S_3 = S_1 + iRow * N;

				// softmax 1) find max
				float max = *S_3;
				float val;
				for (jCol = 0; jCol < N; jCol++){
					val = *(S_3 + jCol);
					
					max = (max > val) ? max : val;
				}

				// softmax 2) sumExp among all elements
				float sumExp = 0.0f;
				for (jCol = 0; jCol < N; jCol++){
					val = *(S_3 + jCol);
					
					sumExp += expf(val - max);
				}
				
				// printf("iRow: %d max: %f sumExp: %f \n", iRow, max, sumExp);
				// softmax 3) for each element, divide by sumExp
				for (jCol = 0; jCol < N; jCol++){
					val = *(S_3 + jCol);
					
					*(S_3 + jCol) = expf(val - max) / sumExp;
					// printf("iRow: %d, jCol: %d, Softmax(S): %f \n", iRow, jCol, *(S_3 + jCol));
				}
			}
			
			// Step 3: softmax output * V
			for (iRow = 0; iRow < N; iRow++){
				for (id = 0; id < d; id++){
					float* S_4 = S_1 + iRow * N;
					float* V_4 = V_1 + id;
					float* O_4 = O_1 + iRow * d + id;

					float accu1 = 0.0f;
					for (iN = 0; iN < N; iN++)
					{
						float S_val = *(S_4 + iN);
						float V_val = *(V_4 + iN * d);
						accu1 += S_val * V_val;
					}

					// printf("iRow: %d, id: %d, result idx: %d, Softmax(S) * V: %f \n", iRow, id, iRow * N + id, accu1);

					*O_4 = accu1;
				}
			}
		}
	}

	return;
}


void flash_att_merge_kernel(float* Q,
							float* K,
							float* V,
							float* O,
							float* S,
							int nBatch,
							int nHead,
							int N,
							int d) 
{
	int iBatch, iHead, iN, id;
	int iRow, jCol;
	
	for (iBatch = 0; iBatch < nBatch; iBatch++){
		for (iHead = 0; iHead < nHead; iHead++){
			// for each batch, length = nHead * N * d
			// for each head, length = N * d
			int Offset = iBatch * nHead * N * d + iHead * N * d;
			float* Q_1 = Q + Offset;
			float* K_1 = K + Offset;
			float* V_1 = V + Offset;
			float* O_1 = O + Offset;
			float* S_1 = S;
			
			for (iRow = 0; iRow < N; iRow++){
				float max_iter = -999999.0f; //-FLT_MAX;
				float max_prev = 0.0f;
				float sumExp_iter = 0.0f;
				float sumExp_prev = 0.0f;
				float Output = 0.0f;
	
				for (jCol = 0; jCol < N; jCol++){
					float* Q_2 = Q_1 + iRow * d;
					float* K_2 = K_1 + jCol * d;
					float* S_2 = S_1 + iRow * N + jCol;

					// Step 1: Q * K_T
					float accu = 0.0f;
					for (id = 0; id < d; id++)
					{
						float Q_val = *(Q_2 + id);
						float K_val = *(K_2 + id);
						accu += Q_val * K_val;
					}
					
					// Step 2: Softmax
					max_iter = (max_iter > accu) ? max_iter : accu;
					float exp_diff = expf(max_prev - max_iter);
					float exp_curr = expf(accu - max_iter);

					sumExp_iter = exp_diff * sumExp_prev + exp_curr;

					// Step 3: Softmax output * V
					float k_i = exp_curr / sumExp_iter;
					
					for (id = 0; id < d; id++)
					{
						float output = *(O_1 + iRow * d + id);
						float V_val = *(V_1 + jCol * d + id);
						
						output = (output * exp_diff * sumExp_prev) / sumExp_iter;

						V_val = output + k_i * V_val;
						*(O_1 + iRow * d + id) = V_val;
					}
					
					// update max_prev, sumExp_prev
					max_prev = max_iter;
					sumExp_prev = sumExp_iter;
				}
			}
		}
	}

	return;
}

void flash_att_merge_block_kernel(float* Q,
							float* K,
							float* V,
							float* O,
							float* S,
							int nBatch,
							int nHead,
							int N,
							int d,
							int Br,
							int Bc)
{
	int iBatch, iHead, iN, id;
	int iRow, jCol;
	const int Tc = N / Bc;
	const int Tr = N / Br;
	int i, j;
	float S_ij[Br][Bc]; // used to store block tile Q * K_T
	float max_S[Br]; // used to store max value of each row in block tile
	float l_expsum[Br]; // used to store exp sum value of each row in block tile
	float O_ij[Br][d]; // used to store the output per block


	for (iBatch = 0; iBatch < nBatch; iBatch++){
		for (iHead = 0; iHead < nHead; iHead++){
			// for each batch, length = nHead * N * d
			// for each head, length = N * d
			int Offset = iBatch * nHead * N * d + iHead * N * d;
			float* Q_0 = Q + Offset;
			float* K_0 = K + Offset;
			float* V_0 = V + Offset;
			float* O_0 = O + Offset;
			float* S_0 = S;
			
			// Initialization
			memset(&S_ij[0][0], 0.0, Br * Bc * sizeof(float));
	
			
			for (i = 0; i < Tr; i++){
				float* Q_1 = Q_0 + i * Br * d;
				float* O_1 = O_0 + i * Br * d;

				float* S_1 = S;
				
				memset(&l_expsum[0], 0.0, Br * sizeof(float));

				for (int iMax = 0; iMax < Br; iMax++){
					max_S[iMax] = -999999.0f;
				}
#if 0
				memset(&O_ij[0][0], 0.0, Br * d * sizeof(float));
#endif
				
				for (j = 0; j < Tc; j++){
					float* K_1 = K_0 + j * Bc * d;
					float* V_1 = V_0 + j * Bc * d;
#if 1
					memset(&O_ij[0][0], 0.0, Br * d * sizeof(float));
#endif

					for (iRow = 0; iRow < Br; iRow++){
						float max_iter = -999999.0f; //-FLT_MAX;
						float max_prev = max_S[iRow];
						float sumExp_iter = 0.0f;
						float sumExp_prev = l_expsum[iRow];
						float Output = 0.0f;
						
						for (jCol = 0; jCol < Bc; jCol++){
							float* Q_2 = Q_1 + iRow * d;
							float* K_2 = K_1 + jCol * d;
							float* S_2 = S_1 + iRow * N + jCol;

							// Step 1: Q * K_T
							float accu = 0.0f;
							for (id = 0; id < d; id++)
							{
								float Q_val = *(Q_2 + id);
								float K_val = *(K_2 + id);
								accu += Q_val * K_val;
							}
										
							max_iter = (max_iter > accu) ? max_iter : accu;
							S_ij[iRow][jCol] = accu;
						}
						
						float exp_curr = 0.0f;
						max_iter = (max_iter > max_prev) ? max_iter : max_prev;
						for (jCol = 0; jCol < Bc; jCol++){
							// Step 2: sum exp per block
							S_ij[iRow][jCol] = expf(S_ij[iRow][jCol] - max_iter);
							exp_curr += S_ij[iRow][jCol];
						}
							
						float exp_diff = expf(max_prev - max_iter);
						
						sumExp_iter = exp_diff * sumExp_prev + exp_curr;

#if 0							
						for (id = 0; id < d; id++)
						{
							float output = O_ij[iRow][id];							
							float temp = (output * exp_diff * sumExp_prev) / sumExp_iter;
							float sum0 = 0.0f;
							
							for (jCol = 0; jCol < Bc; jCol++){
							// Step 3: Softmax output * V for the current block
								float k_i = S_ij[iRow][jCol] / sumExp_iter;
								float V_val = *(V_1 + jCol * d + id);
								
								sum0 += k_i * V_val;
								
							}
							
							O_ij[iRow][id] = temp + sum0;					
						}
#else
						for (jCol = 0; jCol < Bc; jCol++){
							// Step 3: Softmax output * V for the current block
							float k_i = S_ij[iRow][jCol] / sumExp_iter;

							for (id = 0; id < d; id++)
							{
								float V_val = *(V_1 + jCol * d + id);
								
								O_ij[iRow][id] += k_i * V_val;
							}					
						}
						
						// Step 4: add result of current block to final output	
						for (id = 0; id < d; id++)
						{
							float output = *(O_1 + iRow * d + id);							
							float temp = (output * exp_diff * sumExp_prev) / sumExp_iter;

							output = temp + O_ij[iRow][id];
							 
							*(O_1 + iRow * d + id) = output;
						}
#endif
						// update max_prev, sumExp_prev
						max_S[iRow] = max_iter;
						l_expsum[iRow] = sumExp_iter;

					}
				}
				
#if 0
				for (iRow = 0; iRow < Br; iRow++){
					for (id = 0; id < d; id++)
					{						 
						*(O_1 + iRow * d + id) = O_ij[iRow][id];
					}	
				}
#endif
			}
		}
	}

	return;
}

__global__ void flash_att_merge_block_gpu_kernel(float* Q,
												float* K,
												float* V,
												float* O,
												int N,
												int d,
												int Br,
												int Bc)
{
	int i, j, k;
	int iRow, jCol, id;

	// inside a block
	int xIdx = threadIdx.x;
	int yIdx = threadIdx.y;
	// block shape
	int xLen = blockDim.x;
	int yLen = blockDim.y;
	// block index
	int Block_X_Idx = blockIdx.x;
	int Block_Y_Idx = blockIdx.y; // 0
	
	int tid_x = yIdx * xLen + xIdx;
	//int tid_y = xIdx * yLen + yIdx;
	
	// printf("xIdx: %d yIdx: %d xLen: %d yLen: %d tid_x: %d \n", xIdx, yIdx, xLen, yLen, tid_x);
	
	const int Tc = N / Bc;
	const int Tr = N / Br;

	int BlockSize = xLen * yLen;
	
	extern __shared__ float sram[];

	// shared memory
	float *sQ = sram; // shared memory of Q: Br * d * 4
	float *O_ij = sQ + Br * d; // shared memory, used to store the output per block： br * d
	
	float *sK = O_ij + Br * d; // shared memory of K: Bc * d * 4
	float *sV = sK + Bc * d; // shared memory of V: Bc * d * 4
	float *S_ij = sV + Bc * d; // shared memory, used to store block tile Q * K_T: br x bc

	float *max_S = S_ij + Br * Bc;  // shared memory, used to store max value of each row in block tile: Br
	float *l_expsum = max_S + Br; // shared memory, used to store exp sum value of each row in block tile: Br

	int numQ_perThread = Br * d / BlockSize;
	int numKV_perThread = Bc * d / BlockSize;
	int numS_perThread = Bc * Br / BlockSize;
	int iData;
	//printf("Bc: %d, Br: %d, BlockSize: %d, numS_perThread: %d \n", Bc, Br, BlockSize, numS_perThread);

	int iHead = blockIdx.x;
	int iBatch = blockIdx.y;
	int nHead = gridDim.x;
	int nBatch = gridDim.y;
	
	//printf("tidx: %d iHead: %d, iBatch: %d, nHead: %d, nBatch: %d \n", tid_x, iHead, iBatch, nHead, nBatch);
	int Offset = iBatch * nHead * N * d + iHead * N * d;
	float* Q_0 = Q + Offset;
	float* K_0 = K + Offset;
	float* V_0 = V + Offset;
	float* O_0 = O + Offset;
	float sumExp_iter;

#if 1
	for (i = 0; i < Tr; i++){
		// Initialization
		if (tid_x < Br){
			l_expsum[tid_x] = 0.0f;
			max_S[tid_x] = -999999.0f;
		}

		for (iData = 0; iData < numQ_perThread; iData++){
			int threadStep = iData * BlockSize;
			
			O_ij[tid_x + threadStep] = 0.0f;
		}
		
		// load sQ
		int blockTileTr = Br * d * i;

		for (iData = 0; iData < numQ_perThread; iData++){
			int threadStep = iData * BlockSize;
			
			sQ[tid_x + threadStep] = *(Q_0 + tid_x + threadStep + blockTileTr);
		}

		//printf("tid_x: %d, Tc: %d, Tr: %d numS_perThread: %d \n", tid_x, Tc, Tr, numS_perThread);
		for (j = 0; j < Tc; j++){
			// load sK, sV
			int blockTileTc = Bc * d * j;
			for (iData = 0; iData < numKV_perThread; iData++){
				int threadStep = iData * BlockSize;
				
				sK[tid_x + threadStep] = *(K_0 + tid_x + threadStep + blockTileTc);
			}
			
			for (iData = 0; iData < numKV_perThread; iData++){
				int threadStep = iData * BlockSize;
				
				sV[tid_x + threadStep] = *(V_0 + tid_x + threadStep + blockTileTc);
			}
			
			__syncthreads();
			
			int S_idx;
			for (S_idx = 0; S_idx < numS_perThread; S_idx++)
			{
				// index (iRow, jCol) inside S_ij (Q * K_T output) to be calculated in current thread.
				iRow = (S_idx * BlockSize + tid_x) / Bc;
				jCol = (S_idx * BlockSize + tid_x) % Bc;
							
				float* Q_2 = sQ + iRow * d;
				float* K_2 = sK + jCol * d;

				// Step 1: Q * K_T
				float accu = 0.0f;
				for (id = 0; id < d; id++)
				{
					float Q_val = *(Q_2 + id);
					float K_val = *(K_2 + id);
					accu += Q_val * K_val;
				}
				
				//if (tid_x <= 1 && S_idx == 0 && i == 0){
				//	printf("tid_x: %d j: %d S_idx: %d accu: %f r/c: %d %d Q/K: %f %f \n", tid_x, j, S_idx, accu, iRow, jCol, *Q_2, *K_2);
				// }
				S_ij[iRow * Bc + jCol] = accu;
			}
			
			__syncthreads();
			
			if (tid_x < Br){
				float max_iter = -999999.0f; //-FLT_MAX;
				float exp_curr = 0.0f;
				float max_prev = max_S[tid_x];
				float sumExp_prev = l_expsum[tid_x];

				for (jCol = 0; jCol < Bc; jCol++){
				// Step 2a: find max per row in the block br x bc
					float accu = S_ij[tid_x * Bc + jCol];
					max_iter = (max_iter > accu) ? max_iter : accu;
				}
				
				max_iter = (max_iter > max_prev) ? max_iter : max_prev;

				for (jCol = 0; jCol < Bc; jCol++){
				// Step 2b: sum exp per row in the block br x bc
					S_ij[tid_x * Bc + jCol] = __expf(S_ij[tid_x * Bc + jCol] - max_iter);
					exp_curr += S_ij[tid_x * Bc + jCol];
				}

				float exp_diff = __expf(max_prev - max_iter);
				sumExp_iter = exp_diff * sumExp_prev + exp_curr;
				
				// update O_ij
				for (id = 0; id < d; id++){
					float output = O_ij[tid_x * d + id];							
					O_ij[tid_x * d + id] = (output * exp_diff * sumExp_prev) / sumExp_iter;
				}
				
				// update max_prev, sumExp_prev
				max_S[tid_x] = max_iter;
				l_expsum[tid_x] = sumExp_iter;
			}
			
			__syncthreads();

			int O_idx;
			for (O_idx = 0; O_idx < numQ_perThread; O_idx++)
			{
				// index (iRow, id) inside output (P * V output {Br x d} ) to be calculated in current thread.
				iRow = (O_idx * BlockSize + tid_x) / d;
				id = (O_idx * BlockSize + tid_x) % d;
				float sum0 = 0.0f;

				// Step 3: Softmax output * V of the current block for the current thread
				for (int iO = 0; iO < Bc; iO++){
					float V_val = *(sV + iO * d + id);
					float k_i = S_ij[iRow * Bc + iO] / l_expsum[iRow];

					sum0 += k_i * V_val;
				}
				
			
				O_ij[iRow * d + id] += sum0;						
			}
			
			__syncthreads();
		}

		
		// load O_ij to the final reuslt O
		for (iData = 0; iData < numQ_perThread; iData++){
			int threadStep = iData * BlockSize;
			
			*(O_0 + tid_x + threadStep + blockTileTr) = O_ij[tid_x + threadStep];
		}
		
		__syncthreads();
	}
#endif	

	return;
}


int main(int argc, char **argv)
{
    printf("> %s Starting...\n", argv[0]);
	
    // GPT-2 parameters
    int nBatch = 32; // number of batchs
	int nHead = 12; // number of heads
	int HeadDim = 64; // head dimension: 768 / 12 heads
	int nTokens = 1024; // number of tokens in a batch
	
    float *Q;
	float *K;
	float *V;
	float *O_base; // output for reference
	float *O; // output
	float *S; // temporary result to storce Q * K_T
	
	int QKV_Size = nBatch * nHead * HeadDim * nTokens; // 24M
	int QK_T_Size = nTokens * nTokens; // 1MB
	printf("QKV size: %d \n", QKV_Size);
		
	Mimo64_alloc_host_mem((void **)&Q, QKV_Size * sizeof(float)); // 96MB
	Mimo64_alloc_host_mem((void **)&K, QKV_Size * sizeof(float));
	Mimo64_alloc_host_mem((void **)&V, QKV_Size * sizeof(float)); 
	Mimo64_alloc_host_mem((void **)&O_base, QKV_Size * sizeof(float));
	Mimo64_alloc_host_mem((void **)&O, QKV_Size * sizeof(float));
	Mimo64_alloc_host_mem((void **)&S, QK_T_Size * sizeof(float)); // 4MB 


	float *d_Q;
	float *d_K;
	float *d_V;
	float *d_O;

	Mimo64_alloc_device_mem((void **)&d_Q, QKV_Size * sizeof(float));
	Mimo64_alloc_device_mem((void **)&d_K, QKV_Size * sizeof(float));
	Mimo64_alloc_device_mem((void **)&d_V, QKV_Size * sizeof(float));
	Mimo64_alloc_device_mem((void **)&d_O, QKV_Size * sizeof(float));

    memset(O, 0, QKV_Size * sizeof(float));
    memset(O_base,  0, QKV_Size * sizeof(float));

 	initialData_f32(Q, QKV_Size);
 	initialData_f32(K, QKV_Size);
 	initialData_f32(V, QKV_Size);
	
	int Br = 16;
	int Bc = 32;

#if 1
	long t_start = useconds();
	//printf("V: %f %f \n", V[0], V[1]);
	flash_att_naive_kernel(Q, K, V, O_base, S, nBatch, nHead, nTokens, HeadDim);
	//printf("O_base: %f %f \n", O_base[0], O_base[1]);
	
	long t_end = useconds();
	printf("flash_att_naive_kernel() costs %ld us \n", (t_end - t_start) );
#endif

#if 1
	long t_start1 = useconds();
	//printf("V: %f %f \n", V[0], V[1]);
	flash_att_merge_kernel(Q, K, V, O, S, nBatch, nHead, nTokens, HeadDim);
	//printf("O: %f %f \n", O[0], O[1]);
	
	long t_end1 = useconds();
	printf("flash_att_merge_kernel() costs %ld us \n", (t_end1 - t_start1) );

	checkResult(O_base, O, QKV_Size);	

    memset(O, 0, QKV_Size * sizeof(float));
	long t_start2 = useconds();
	//printf("V: %f %f \n", V[0], V[1]);
	flash_att_merge_block_kernel(Q, K, V, O, S, nBatch, nHead, nTokens, HeadDim, Br, Bc);
	//printf("O: %f %f \n", O[0], O[1]);
	
	long t_end2 = useconds();
	printf("flash_att_merge_block_kernel() costs %ld us \n", (t_end2 - t_start2) );

	checkResult(O_base, O, QKV_Size);
#endif

#if 1
	float kernel_time;
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

	CHECK(hipMemcpy(d_Q, Q, QKV_Size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_K, K, QKV_Size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_V, V, QKV_Size * sizeof(float), hipMemcpyHostToDevice));
    
	// Calculate SRAM size needed per block
    const int sram_size = (2 * Bc * HeadDim * sizeof(float)) + (2 * Br * HeadDim * sizeof(float)) 
						+ (Bc * Br * sizeof(float)) + (2 * Br * HeadDim * sizeof(float));
						
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);
	
	int skipKernel = 0;

	if (max_sram_size < sram_size){
		skipKernel = 1;
		printf("Your request memory is larger than system volume, please input another Br/Bc combination! \n");
	}
	int block_x = 512;
	int block_y = 1;
	
	int numThreads = block_x * block_y;
	int align0 = Bc * HeadDim;
	int align1 = Br * HeadDim;
	int align2 = Bc * Br;
		
	if (align0 % numThreads != 0){
		skipKernel = 1;
		printf("Num of Threads must align with Bc * HeadDim \n");
	}		
		
	if (align1 % numThreads != 0){
		skipKernel = 1;
		printf("Num of Threads must align with Br * HeadDim \n");
	}		

	if (align2 % numThreads != 0){
		skipKernel = 1;
		printf("Num of Threads must align with Bc * Br \n");
	}

	if (skipKernel == 0){
		dim3 grid(nHead, nBatch);
		dim3 block(block_x, block_y);

		CHECK(hipEventRecord(start, 0));

		flash_att_merge_block_gpu_kernel<<<grid, block, sram_size>>>(d_Q, d_K, d_V, d_O, nTokens, HeadDim, Br, Bc);

		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		CHECK(hipEventElapsedTime(&kernel_time, start, stop));

		CHECK(hipMemcpy(O, d_O, QKV_Size * sizeof(float), hipMemcpyDeviceToHost));

		printf("flash_att_merge_block_gpu_kernel() costs %ld us \n", (long)(kernel_time * 1000.0f));

		checkResult(O_base, O, QKV_Size);	
	}
	
#endif

	Mimo64_free_host_mem(Q);
	Mimo64_free_host_mem(K);
	Mimo64_free_host_mem(V);
	Mimo64_free_host_mem(O);
	Mimo64_free_host_mem(O_base);
	
	Mimo64_free_device_mem(d_Q);
	Mimo64_free_device_mem(d_K);
	Mimo64_free_device_mem(d_V);
	Mimo64_free_device_mem(d_O);

	return 0;
}