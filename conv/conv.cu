#include "hip/hip_runtime.h"
#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "./common.h"
#include <stdint.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <math.h>

#define DOUBLE(pointer) (reinterpret_cast<double*>(&(pointer))[0])
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define float16_t half
//#define printf(...);
#define IM2COL_BERVER 0

void Mimo64_alloc_host_mem(void** host_ptr_addr, size_t size)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    err = hipHostMalloc((void **)host_ptr_addr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate host memory (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	return;
}

void Mimo64_free_host_mem(void* host_ptr)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    if (host_ptr != NULL)
    {
        err = hipHostFree(host_ptr);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to free host memory (error code %s)!\n",
                    hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
	
	return;
}

void Mimo64_alloc_device_mem(void** dev_ptr_addr, size_t size)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
  
	err = hipMalloc((void**)dev_ptr_addr, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate cuda device mem (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	return;
}

void Mimo64_free_device_mem(void* dev_ptr)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    if (dev_ptr != NULL){
		err = hipFree(dev_ptr);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to free cuda device mem (error code %s)!\n",
                    hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	}

	return;
}


void initialData_f32(float *ip, int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)rand() / (float)RAND_MAX;
		//printf("val[%d]: %d \n", i, ip[i]);
    }

	return;
}


void initialData_Y_f32(float *ip, int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
		if (i < size)
			ip[i] = (float) 1.0f; //(rand() & 0xFF);
		//printf("val[%d]: %d \n", i, ip[i]);
		else
			ip[i] = (float)0.0; //(rand() & 0xFF);
		//printf("val[%d]: %d \n", i, ip[i]);
    }

	return;
}

void Mimo64_init_device_const_mem()
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

 

	return;
}


void float2half(float16_t *dst, float *src, int nElem){
	for (int i = 0; i < nElem; i++){
		dst[i] = __float2half(src[i]);
	}
	
	return;
}

hipStream_t *streams;

void Mimo64_createStreams(int numOfStreams){

	streams = (hipStream_t *)malloc(numOfStreams * sizeof(hipStream_t));

	for (int i = 0; i < numOfStreams; i++)
		hipStreamCreate(&streams[i]);

	return;
}


void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-2;
    bool match = 1;
	
//	printf("hello world. \n");
//		printf("host: %f gpu: %f \n", hostRef[0], gpuRef[0]);
    for (int i = 0; i < N; i++)
    {
		//printf("i: %d host: %f gpu: %f \n", i, hostRef[i], gpuRef[i]);
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %f gpu %f at %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");
}



void conv_naive_kernel(
	float *input,      // input image: [batch_size][in_channels][height][width]
    float *kernel,     // conv kernel: [out_channels][in_channels][kernel_h][kernel_w]
    float *bias,       // bias： [out_channels] (可为NULL)
    float *output,     // output: [batch_size][out_channels][out_h][out_w]
    int batch_size,    // batch
    int in_channels,   // input channel: RGB
    int out_channels,  // output channel: num of kernels
    int height,        // image height
    int width,         // image width
    int kernel_h,      // kernel height
    int kernel_w,      // kernel width
    int stride_h,      // stride height
    int stride_w,      // stride width
    int padding_h,     // padding height
    int padding_w      // padding width
) 
{
	int iBatch, iKernel, iChannel;
	int iRow, jCol;
	
	// output size in one batch one kernel
	int out_height = (height + 2 * padding_h - kernel_h) / stride_h + 1;
    int out_width = (width + 2 * padding_w - kernel_w) / stride_w + 1;
  
	// input size per batch
 	int numPixelperChannel = height * width; // input image
	int numPixelperBatch = numPixelperChannel * in_channels;
	
	int numEleperKernelperChannel = kernel_h * kernel_w; // input kernel
	int numEleperKernel = numEleperKernelperChannel * in_channels;
	
	int numElePerOutKernel = out_height * out_width;
	int numElePerOutBatch = out_channels *  numElePerOutKernel; // output

	// output initialization
	int output_size = batch_size * numElePerOutBatch;
    for (int i = 0; i < output_size; i++) {
        output[i] = 0.0f;
    }
	

	for (iBatch = 0; iBatch < batch_size; iBatch++){
		// for each kernel
		for (iKernel = 0; iKernel < out_channels; iKernel++){
			// locate to given img
			float *imgIn = input + iBatch * numPixelperBatch;
			float *kerIn = kernel + iKernel * numEleperKernel;
			float *convOut = output + iBatch * numElePerOutBatch + iKernel * numElePerOutKernel;
			
			// for each RGB input channels
			for (iChannel = 0; iChannel < in_channels; iChannel++){
				float *imgIn_Ch = imgIn + iChannel * numPixelperChannel;
				float *kerIn_Ch = kerIn + iChannel * numEleperKernelperChannel;
				
				// for each element in output 
				for (int oh = 0; oh < out_height; oh++) {
                    for (int ow = 0; ow < out_width; ow++) {
                        // start pos in the given imag
                        int start_h = oh * stride_h - padding_h;
                        int start_w = ow * stride_w - padding_w;
                        int output_idx = oh * out_width + ow;
		
								
						// every output element need loop conv kernel
						for (int kh = 0; kh < kernel_h; kh++) {
                            for (int kw = 0; kw < kernel_w; kw++) {
                                int ih = start_h + kh;
                                int iw = start_w + kw;
                                
                                // check if it is padding
                                if (ih >= 0 && ih < height && iw >= 0 && iw < width) {
                                    int input_idx = ih * width + iw;
                                    int kernel_idx = kh * kernel_w + kw;
                                    
                                    // conv
                                    convOut[output_idx] += imgIn_Ch[input_idx] * kerIn_Ch[kernel_idx];
                                } else {
									// if padding 0, then do nothing
									
								}
							}
                        }
					}
				}
			}
			
			// add bias
			float bias_val = bias ? bias[iKernel] : 0.0f;
			
			//printf("bias_val : %f \n", bias_val);
			if (bias) {
                for (int oh = 0; oh < out_height; oh++) {
                    for (int ow = 0; ow < out_width; ow++) {
                        int output_idx = oh * out_width + ow;
                        convOut[output_idx] += bias_val;
                    }
                }
            }
			
		}
	}
	

	return;
}


/**
 * 
 * im2col - covert input image data to col matrix
 * 
 */
void im2col(
	const float* input,
	float* data_col,
	int channels,
	int height,
	int width,
	int kernel_h,
	int kernel_w,
	int pad_h, 
	int pad_w,
	int stride_h, 
	int stride_w)
{
    int out_h = (height + 2 * pad_h - kernel_h) / stride_h + 1;
    int out_w = (width + 2 * pad_w - kernel_w) / stride_w + 1;
    
    // col matrix size
    int col_rows = channels * kernel_h * kernel_w;
    int col_cols = out_h * out_w;
    
	//printf("col_rows: %d, col_cols: %d \n", col_rows, col_cols);
	
	// loop every element in output col matrix
	for (int oh = 0; oh < out_h; oh++) {
		for (int ow = 0; ow < out_w; ow++) {
            // start pos in the given imag
			int start_h = oh * stride_h - pad_h;
			int start_w = ow * stride_w - pad_w;
						
			// col idx in output col matrix
			int col_idx = oh * out_w + ow;
			
			// every output element need loop conv kernel
			for (int kh = 0; kh < kernel_h; kh++) {
				for (int kw = 0; kw < kernel_w; kw++) {
					// actual idx in the orig imag
					int ih = start_h + kh;
					int iw = start_w + kw;
					
					// every channel (RGB)
					for (int c = 0; c < channels; c++) {
						// row idx in output col matrix
						int row_idx = c * (kernel_h * kernel_w) + kh * kernel_w + kw;
			
						
						// input index
						int input_idx = c * (height * width) + ih * width + iw;
						
                        // check if it is padding
						if (ih >= 0 && ih < height && iw >= 0 && iw < width) {
							data_col[row_idx * col_cols + col_idx] = input[input_idx];
						} else {
							data_col[row_idx * col_cols + col_idx] = 0.0f;
						}
					}
				}
			}
		}
	}
	
	return;
}


/**
 * 
 * matmul_add_bias - coverted input image matrix * kernel + bias
 * 
 */
void matmul_add_bias(
	const float* data_col, 
	const float* kernel, 
	const float* bias, 
	float* output,
    int out_channels, 
	int col_rows, 
	int col_cols, 
	int out_h, 
	int out_w) 
{
    // for each filter kernel
    for (int oc = 0; oc < out_channels; oc++) {
        float bias_val = bias ? bias[oc] : 0.0f;
        
        
        for (int col = 0; col < col_cols; col++) {
            float sum = 0.0f;
            
            // convolution
            for (int row = 0; row < col_rows; row++) {
                sum += data_col[row * col_cols + col] * kernel[oc * col_rows + row];
            }
            
            // bias
            sum += bias_val;
            
            // output index
           // int pos = col % (out_h * out_w);
           // int oh = pos / out_w;
           // int ow = pos % out_w;
            
            int output_idx = oc * (out_h * out_w) + col;
            
            output[output_idx] = sum;
        }
    }
}

#if IM2COL_BERVER == 1    
float im2col_get_pixel(float *im, int height, int width, int channels,
                        int row, int col, int channel, int pad)
{
    row -= pad;
    col -= pad;

    if (row < 0 || col < 0 ||
        row >= height || col >= width) return 0;
    return im[col + width*(row + height*channel)];
}

//From Berkeley Vision's Caffe!
//https://github.com/BVLC/caffe/blob/master/LICENSE
void im2col_cpu(float* data_im,
     int channels,  int height,  int width,
     int ksize,  int stride, int pad, float* data_col) 
{
    int c,h,w;
    int height_col = (height + 2*pad - ksize) / stride + 1;
    int width_col = (width + 2*pad - ksize) / stride + 1;

    int channels_col = channels * ksize * ksize;
    for (c = 0; c < channels_col; ++c) {
        int w_offset = c % ksize;
        int h_offset = (c / ksize) % ksize;
        int c_im = c / ksize / ksize;
        for (h = 0; h < height_col; ++h) {
            for (w = 0; w < width_col; ++w) {
                int im_row = h_offset + h * stride;
                int im_col = w_offset + w * stride;
                int col_index = (c * height_col + h) * width_col + w;
                data_col[col_index] = im2col_get_pixel(data_im, height, width, channels,
                        im_row, im_col, c_im, pad);
            }
        }
    }
}
#endif

void conv_im2col_kernel(	
	float *input,      // input image: [batch_size][in_channels][height][width]
    float *kernel,     // conv kernel: [out_channels][in_channels][kernel_h][kernel_w]
    float *bias,       // bias： [out_channels] (可为NULL)
    float *output,     // output: [batch_size][out_channels][out_h][out_w]
	float *data_col,   // converted input image --> col matrix: [batch_size][kernel height * kernel width * in_channels][out_h * out_w]
    int batch_size,    // batch
    int in_channels,   // input channel: RGB
    int out_channels,  // output channel: num of kernels
    int height,        // image height
    int width,         // image width
    int kernel_h,      // kernel height
    int kernel_w,      // kernel width
    int stride_h,      // stride height
    int stride_w,      // stride width
    int padding_h,     // padding height
    int padding_w      // padding width
){
	int iBatch;
    int out_h = (height + 2 * padding_h - kernel_h) / stride_h + 1;
    int out_w = (width + 2 * padding_w - kernel_w) / stride_w + 1;
    
	// input size per batch
 	int numPixelperChannel = height * width; // input image
	int numPixelperBatch = numPixelperChannel * in_channels;
	int numElePerOutKernel = out_h * out_w;
	int numElePerOutBatch = out_channels *  numElePerOutKernel; // output

	int numColMatrixperBatch = kernel_h * kernel_w * in_channels * out_h * out_w; 
    // converted input image --> col matrix
    int col_rows = in_channels * kernel_h * kernel_w;
    int col_cols = out_h * out_w;
	
	for (iBatch = 0; iBatch < batch_size; iBatch++){
		float *input_batch = input + numPixelperBatch * iBatch;
		float *data_col_batch = data_col + numColMatrixperBatch * iBatch; // actually we can use one batch for intermediate use
		float *output_batch = output + numElePerOutBatch * iBatch;
		
#if IM2COL_BERVER == 1    
		im2col_cpu(input_batch, in_channels,  height,  width, kernel_h,
					stride_h, padding_h, data_col_batch);
#else 
		im2col(input_batch, data_col_batch, in_channels, height, width,
			kernel_h, kernel_w, padding_h, padding_w, stride_h, stride_w);
#endif	
	
		matmul_add_bias(data_col_batch, kernel, bias, output_batch, 
                    out_channels, col_rows, col_cols, out_h, out_w);
 
	}

	return;
}

__constant__ float d_kernel_const[32 * 3 * 5 * 5];

__global__ void conv_im2col_gpu_kernel(
					float *input,      // input image: [batch_size][in_channels][height][width]
					float *bias,       // bias： [out_channels] (可为NULL)
					float *output,     // output: [batch_size][out_channels][out_h][out_w]
					int batch_size,    // batch
					int in_channels,   // input channel: RGB
					int out_channels,  // output channel: num of kernels
					int height,        // image height
					int width,         // image width
					int kernel_h,      // kernel height
					int kernel_w,      // kernel width
					int stride_h,      // stride height
					int stride_w,      // stride width
					int padding_h,     // padding height
					int padding_w      // padding width)
) {
	int i, j, k;
	
	// inside a block
	int tid_x = threadIdx.x;
	// block shape
	int xLen = blockDim.x;
	int yLen = blockDim.y;
	// block index
	int bx = blockIdx.x;
	int by = blockIdx.y; // batch index
	
	extern __shared__ float sram[];

	// shared memory
	float *sInImgCh0 = sram; // shared memory of input imag: width * kernel_h
	float *sInImgCh1 = sInImgCh0 + width * kernel_h; // shared memory of input imag: width * kernel_h
	float *sInImgCh2 = sInImgCh1 + width * kernel_h; // shared memory of input imag: width * kernel_h
	float *sCvtColMatrix = sInImgCh2 + width * kernel_h; // shared memory of input imag: 128 * in_channels * kernel_h * kernel_w

	// step 1: load img from global to shared
	int start_row = bx - padding_h;
	int s_start_row = (start_row >= 0) ? 0 : (-start_row);
	int g_start_row = (start_row >= 0) ? start_row : 0;

	int num_row = (start_row >= 0) ? kernel_h : (kernel_h + start_row);
	
	int num_row_tail = (height - kernel_h) - start_row;
	num_row = (num_row_tail > 0) ? num_row : (kernel_h + num_row_tail);

	// if (tid_x == 0)
	//	printf("bx: %d, num_row: %d, s_start_row: %d \n", bx, num_row, s_start_row);

	// input size per batch
 	int numPixelperChannel = height * width; // input image
	int numPixelperBatch = numPixelperChannel * in_channels;

	float *gInImg = input + by * numPixelperBatch;
	float *gInImgCh0 = gInImg;
	float *gInImgCh1 = gInImgCh0 + numPixelperChannel;
	float *gInImgCh2 = gInImgCh1 + numPixelperChannel;

	// set to 0, 5 * 128 is less than 1024
	if (tid_x < width * kernel_h){
		sInImgCh0[tid_x]  = 0.0f;
		sInImgCh1[tid_x]  = 0.0f;
		sInImgCh2[tid_x]  = 0.0f;
	}

	if (tid_x < width * num_row){
		sInImgCh0[width * s_start_row + tid_x]  = gInImgCh0[g_start_row * width + tid_x];
		sInImgCh1[width * s_start_row + tid_x]  = gInImgCh1[g_start_row * width + tid_x];
		sInImgCh2[width * s_start_row + tid_x]  = gInImgCh2[g_start_row * width + tid_x];
	}
	
	__syncthreads();

	
	// step 2: im2col - need to be optimized
    int out_h = (height + 2 * padding_h - kernel_h) / stride_h + 1;
    int out_w = (width + 2 * padding_w - kernel_w) / stride_w + 1;
 
	int col_rows = in_channels * kernel_h * kernel_w;
	int col_cols = out_w;

	int numElePerOutKernel = out_h * out_w;
	int numElePerOutBatch = out_channels * numElePerOutKernel; // output

	float conv_5x5[5*5] = {0.0f};
	float *startImg;
	float *tempCvtColMatrix;
	
	if (tid_x < 384)
	{
		startImg = sInImgCh2;
		tempCvtColMatrix = sCvtColMatrix + 2 * kernel_h * kernel_w * col_cols;
	}
	
	if (tid_x < 256)
	{
		startImg = sInImgCh1;
		tempCvtColMatrix = sCvtColMatrix + kernel_h * kernel_w * col_cols;
	}
	
	if (tid_x < 128){
		startImg = sInImgCh0;
		tempCvtColMatrix = sCvtColMatrix;
	}
	
	if (tid_x < 384){
		int tid_rem = tid_x % 128;
		int start_conv = tid_rem - padding_w;
		
		int s_start_col = (start_conv >= 0) ? 0 : (-start_conv);
		int s_end_col = (start_conv > (width - kernel_w)) ? (width - start_conv) : 5;

		//if (tid_x < 128 && bx == 0){
			//printf("tid_x: %d, s_start_col: %d, s_end_col: %d \n", tid_x, s_start_col, s_end_col);
		//}
		
		int s_start_col_img = (start_conv >= 0) ? start_conv : 0;

		for (int conv_i = 0; conv_i < 5; conv_i ++)
		{
			for (int conv_j = s_start_col; conv_j < s_end_col; conv_j ++)
			{
				conv_5x5[conv_i * 5 + conv_j] = startImg[ conv_i * 128 + s_start_col_img + conv_j - s_start_col ];
			}
		}
		
		// write into converted col matrix
		for (int conv_i = 0; conv_i < 25; conv_i++)
		{
			tempCvtColMatrix[conv_i * col_cols + tid_rem] = conv_5x5[conv_i];
		}
	}

	__syncthreads();

/*
		if (tid_x == 0 && bx == 0 && by == 0)
		{
			for (int conv_i = 0; conv_i < 25; conv_i++)
				printf("conv_i: %d val: %f \n", conv_i, conv_5x5[conv_i]);
		}

	__syncthreads();
*/
#if 1
	// step 3: matmul, each thread calculate 2x2 = 4 elements
	int rowIdx = tid_x / 64;
	int colIdx = tid_x % 64;
	int iRow, iCol;
	
	float res[2][2];
	rowIdx *= 2;
	colIdx *= 2;
	
	for (iRow = rowIdx; iRow < (rowIdx + 2); iRow++){
		for (iCol = colIdx; iCol < (colIdx + 2); iCol++){

			float bias_val = bias ? bias[iRow] : 0.0f;
        
			float sum = 0.0f;

			// convolution
			for (int kIdx = 0; kIdx < col_rows; kIdx++) {				
				sum += sCvtColMatrix[kIdx * col_cols + iCol] * d_kernel_const[iRow * col_rows + kIdx];
			}
			
			// bias
			sum += bias_val;
			
			res[iRow - rowIdx][iCol - colIdx] = sum;
			
		}
	}

	__syncthreads();
	
	
	float *output_block = output + by * numElePerOutBatch + rowIdx * numElePerOutKernel + bx * out_h + colIdx;
	*(output_block) = res[0][0];
	*(output_block + 1) = res[0][1];

	float *output_block1 = output + by * numElePerOutBatch + (rowIdx + 1) * numElePerOutKernel + bx * out_h + colIdx;
	*(output_block1) = res[1][0];
	*(output_block1 + 1) = res[1][1];
#endif

	return;
}


int main(int argc, char **argv)
{
    printf("> %s Starting...\n", argv[0]);
	
    // conv parameters
    const int nBatch = 32; // number of batchs
	const int imgHeight = 128; // image height
	const int imgWidth = 128; // image width
	const int kernelHeight = 5; // kernel Height
	const int kernelWidth = 5; // kernel Height
	const int stride_h = 1; // stride in height direction
	const int stride_w = 1; // stride in width direction
	const int padding_h = 2; // padding in height direction
	const int padding_w = 2; // paddign in width direction
	const int inChannel = 3; // 3 channels per image
	const int outChannel = 32; // 32 kernels

    float *inImg;
	float *inImgCol;
	float *kernel;
	float *bias;
	float *O_base; // output for reference
	float *O; // output
	float *S; // temporary result 
	
	int out_height = (imgHeight + 2 * padding_h - kernelHeight) / stride_h + 1;
    int out_width = (imgWidth + 2 * padding_w - kernelWidth) / stride_w + 1;
    // converted input image --> col matrix
	int inImgColSize = nBatch * out_height * out_width * inChannel * kernelHeight * kernelWidth;
	
	int imgSize = nBatch * inChannel * imgHeight * imgHeight; // 1.5MB
	const int kernelSize = outChannel * inChannel * kernelHeight * kernelWidth;
	// keep same size with input
	int oSize = nBatch * outChannel * out_height * out_width; // 1MB

	printf("img size: %d kernel size: %d converted col matrix: %d out size: %d \n", imgSize, kernelSize, inImgColSize, oSize);

	Mimo64_alloc_host_mem((void **)&inImg, imgSize * sizeof(float)); // 6MB
	Mimo64_alloc_host_mem((void **)&kernel, kernelSize * sizeof(float));
	Mimo64_alloc_host_mem((void **)&bias, outChannel * sizeof(float));
	Mimo64_alloc_host_mem((void **)&inImgCol, inImgColSize * sizeof(float));


	Mimo64_alloc_host_mem((void **)&O_base, oSize * sizeof(float));
	Mimo64_alloc_host_mem((void **)&O, oSize * sizeof(float));

	float *d_inImg;
	float *d_kernel;
	float *d_bias;
	float *d_O;

	Mimo64_alloc_device_mem((void **)&d_inImg, imgSize * sizeof(float));
	Mimo64_alloc_device_mem((void **)&d_kernel, kernelSize * sizeof(float));
	Mimo64_alloc_device_mem((void **)&d_O, oSize * sizeof(float));
	Mimo64_alloc_device_mem((void **)&d_bias, outChannel * sizeof(float));

    memset(O, 0, oSize * sizeof(float));
    memset(O_base,  0, oSize * sizeof(float));

 	initialData_f32(inImg, imgSize);
 	initialData_f32(kernel, kernelSize);
 	initialData_f32(bias, outChannel);
	

	//bias = NULL;
#if 1
	long t_start = useconds();
	conv_naive_kernel
	(
		inImg,
		kernel,
		bias,
		O_base,
		nBatch,
		inChannel,
		outChannel,
		imgHeight,
		imgWidth,
		kernelHeight,
		kernelWidth,
		stride_h,
		stride_w,
		padding_h,
		padding_w
	);
	//printf("O_base: %f %f \n", O_base[0], O_base[1]);
	
	long t_end = useconds();
	printf("conv_naive_kernel() costs %ld us \n", (t_end - t_start) );
#endif

#if 1
	long t_start1 = useconds();
	
	conv_im2col_kernel
	(
		inImg,
		kernel,
		bias,
		O,
		inImgCol,
		nBatch,
		inChannel,
		outChannel,
		imgHeight,
		imgWidth,
		kernelHeight,
		kernelWidth,
		stride_h,
		stride_w,
		padding_h,
		padding_w
	);
	
	long t_end1 = useconds();
	printf("conv_im2col_kernel() costs %ld us \n", (t_end1 - t_start1) );

	checkResult(O_base, O, oSize);
#endif

#if 1
	float kernel_time;
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));


	CHECK(hipMemcpy(d_inImg, inImg, imgSize * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_const), kernel, kernelSize * sizeof(float), 0, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias, bias, outChannel * sizeof(float), hipMemcpyHostToDevice));
    
	const int numColPerBlk = 128;
	int colBlkSize = numColPerBlk * inChannel * kernelHeight * kernelWidth;
	int inputBlkSize = numColPerBlk * kernelHeight * inChannel;

	// Calculate SRAM size needed per block
    const int sram_size = (colBlkSize * sizeof(float)) /* col matrix block size*/
						+ (inputBlkSize * sizeof(float)) /* input image */;
						
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);
	
	int skipKernel = 0;

	if (max_sram_size < sram_size){
		skipKernel = 1;
		printf("Your request memory is larger than system volume, please input another Br/Bc combination! \n");
	}
	int block_x = 1024;
	int block_y = 1;
	int gridIdx_x = (out_height * out_width) / numColPerBlk;
	//printf("gridIdx_x: %d \n", gridIdx_x);
	
	if (skipKernel == 0){
		dim3 grid(gridIdx_x, nBatch);
		dim3 block(block_x, block_y);

		CHECK(hipEventRecord(start, 0));

		conv_im2col_gpu_kernel<<<grid, block, sram_size>>>(	d_inImg,
															d_bias,
															d_O,
															nBatch,
															inChannel,
															outChannel,
															imgHeight,
															imgWidth,
															kernelHeight,
															kernelWidth,
															stride_h,
															stride_w,
															padding_h,
															padding_w);

		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		CHECK(hipEventElapsedTime(&kernel_time, start, stop));

		CHECK(hipMemcpy(O, d_O, oSize * sizeof(float), hipMemcpyDeviceToHost));

		printf("conv_im2col_block_gpu_kernel() costs %ld us \n", (long)(kernel_time * 1000.0f));

		checkResult(O_base, O, oSize);	
	}
	
#endif





	Mimo64_free_host_mem(inImg);
	Mimo64_free_host_mem(kernel);
	Mimo64_free_host_mem(bias);
	Mimo64_free_host_mem(O);
	Mimo64_free_host_mem(O_base);
	
	Mimo64_free_device_mem(d_bias);
	Mimo64_free_device_mem(d_inImg);
	Mimo64_free_device_mem(d_kernel);
	Mimo64_free_device_mem(d_O);

	return 0;
}