#include "hip/hip_runtime.h"
#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include "./common.h"
#include <stdint.h>
#include <hip/hip_fp16.h>
#include <mma.h>

using namespace nvcuda;

#define float16_t half


void Mimo64_alloc_host_mem(void** host_ptr_addr, size_t size)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    err = hipHostMalloc((void **)host_ptr_addr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate host memory (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	return;
}

void Mimo64_free_host_mem(void* host_ptr)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    if (host_ptr != NULL)
    {
        err = hipHostFree(host_ptr);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to free host memory (error code %s)!\n",
                    hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
	
	return;
}

void Mimo64_alloc_device_mem(void** dev_ptr_addr, size_t size)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
  
	err = hipMalloc((void**)dev_ptr_addr, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate cuda device mem (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	return;
}

void Mimo64_free_device_mem(void* dev_ptr)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    if (dev_ptr != NULL){
		err = hipFree(dev_ptr);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to free cuda device mem (error code %s)!\n",
                    hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
	}

	return;
}


void initialData_f32(float *ip, int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF);
		//printf("val[%d]: %d \n", i, ip[i]);
    }

	return;
}


void initialData_Y_f32(float *ip, int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
		if (i < size)
			ip[i] = (float)(rand() & 0xFF);
		//printf("val[%d]: %d \n", i, ip[i]);
		else
			ip[i] = (float)0.0; //(rand() & 0xFF);
		//printf("val[%d]: %d \n", i, ip[i]);
    }

	return;
}

void Mimo64_init_device_const_mem()
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

 

	return;
}


void float2half(float16_t *dst, float *src, int nElem){
	for (int i = 0; i < nElem; i++){
		dst[i] = __float2half(src[i]);
	}
	
	return;
}

hipStream_t *streams;

void Mimo64_createStreams(int numOfStreams){

	streams = (hipStream_t *)malloc(numOfStreams * sizeof(hipStream_t));

	for (int i = 0; i < numOfStreams; i++)
		hipStreamCreate(&streams[i]);

	return;
}


void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;
	
//	printf("hello world. \n");
//	printf("host: %f gpu: %f \n", hostRef[0], gpuRef[0]);
    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %f gpu %f at %d (block: %d, thread: %d)\n", hostRef[i], gpuRef[i], i, i / (64 * 4), i % (64 * 4));
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");
}




void mimo64_naive_kernel(float *G, float *Y, float *X, int nElem, int nElemPerMatrix)
{
	int numOfloop = nElem / nElemPerMatrix;
	int loopIdx;
	int iRow, iCol, iK;
	float *G0, *Y0, *X0;
	
	for (loopIdx = 0; loopIdx < numOfloop; loopIdx++)
	{
		G0 = G + 64 * 64 * loopIdx;
		Y0 = Y + 64 * nElemPerMatrix * loopIdx;
		X0 = X + 64 * nElemPerMatrix * loopIdx;
		
		// G matrix (64 x 64) x Y matrix (64 x nElemPerMatrix(4))
		for (iRow = 0; iRow < 64; iRow++)
		{
			for (iCol = 0; iCol < nElemPerMatrix; iCol++)
			{
				float accuVal = 0.0f;
				
				for (iK = 0; iK < 64; iK++)
				{
					
					accuVal += G0[iRow * 64 + iK] * Y0[iK * nElemPerMatrix + iCol];	
				}
				
				X0[iRow * nElemPerMatrix + iCol] = accuVal;
			}
		}
		
	}


	return;
}

__global__ void mimo64_naive_gpu_kernel(float *G, float *Y, float *X, int nElem)
{
	int loopIdx;
	int iRow, iCol, iK;
	float *G0, *Y0, *X0;
	int nElemPerMatrix = 4;
	
	// inside a block
	int xIdx = threadIdx.x;
	int yIdx = threadIdx.y;
	// block shape
	int xLen = blockDim.x;
	int yLen = blockDim.y;
	// block index
	int Block_X_Idx = blockIdx.x;
	int Block_Y_Idx = blockIdx.y; // 0
	
	int tid_x = Block_X_Idx * xLen + xIdx;
	int tid_y = Block_Y_Idx * yLen + yIdx;
	
	// printf("xIdx: %d yIdx: %d \n", xIdx, yIdx);

	G0 = G + 64 * 64 * Block_X_Idx;
	Y0 = Y + 64 * nElemPerMatrix * Block_X_Idx;
	X0 = X + 64 * nElemPerMatrix * Block_X_Idx;
	
	iRow = yIdx;
	iCol = xIdx;
	//for (iRow = 0; iRow < 64; iRow++)
	{
		//for (iCol = 0; iCol < nElemPerMatrix; iCol++)
		{
			float accuVal = 0.0f;
			
			for (iK = 0; iK < 64; iK++)
			{
				
				accuVal += G0[iRow * 64 + iK] * Y0[iK * nElemPerMatrix + iCol];
			
			}
			
			X0[iRow * nElemPerMatrix + iCol] = accuVal;
		}
	}
	
	return;
}

template <
    const int BLOCK_SIZE_M,  // height of block of X that each thread calculate, i.e. bm
    const int BLOCK_SIZE_K,  // width of block of G that each thread load into shared memory, i.e. bk
    const int BLOCK_SIZE_N  // width of block of X that each thread calculate i.e. bn
    > 
__global__ void mimo64_block_naive_gpu_kernel(float *G, float *Y, float *X, int nElem)
{
	int i, j, k;
	
	// inside a block
	int xIdx = threadIdx.x; 
	int yIdx = threadIdx.y;
	// block shape
	int xLen = blockDim.x;
	int yLen = blockDim.y;
	// block index
	int Block_X_Idx = blockIdx.x;
	int Block_Y_Idx = blockIdx.y; // 0
	
	int tid_x = Block_X_Idx * xLen + xIdx;
	int tid_y = Block_Y_Idx * yLen + yIdx;

	int xUnit = BLOCK_SIZE_K / xLen; // BLOCK_SIZE_K is multiple of xLen
		
	// shared memory
	__shared__ float Gs[64][BLOCK_SIZE_K]; // shared memory of G: 256 bytes x BLOCK_SIZE_K
	__shared__ float Ys[64][4]; // shared memory of Y: 1 Kbyte
	
	float accu[BLOCK_SIZE_M][BLOCK_SIZE_N];
	float *G0, *Y0, *X0;

	G0 = G + 64 * 64 * Block_X_Idx;
	Y0 = Y + 64 * 4 * Block_X_Idx;
	X0 = X + 64 * 4 * Block_X_Idx;

	// load data from global memory to shared memeory
	// load the whole Y 
	int sLoadIdx;
	float *Gs_0, *Ys_0;
	Gs_0 = &Gs[0][0];
	Ys_0 = &Ys[0][0];
	
	int nElemPerThread = 64 * 4 / (xLen * yLen);

	int currentThreadIdx = yIdx * xLen + xIdx;
	
	for (i = 0; i < nElemPerThread; i++)
	{
		sLoadIdx = currentThreadIdx * nElemPerThread + i;

		Ys_0[sLoadIdx] = Y0[sLoadIdx];
	}
	
	#pragma unroll
	for (i = 0; i < BLOCK_SIZE_M; i++)
	{
	    #pragma unroll
		for (j = 0; j < xUnit; j++)
		{
			int mIdx = yIdx * BLOCK_SIZE_M + i;
			// note: int kIdx = /*xIdx * BLOCK_SIZE_K +*/ j;
			int kIdx = xIdx * xUnit + j; // BLOCK_SIZE_K: 1, 2, 4
			
			sLoadIdx = 64 * mIdx + kIdx;
			
			Gs[mIdx][kIdx] = G0[sLoadIdx];
		}
	}

	// reset the accumulation
	#pragma unroll
	for (i = 0; i < BLOCK_SIZE_M; i++)
	{
		#pragma unroll
		for (j = 0; j < BLOCK_SIZE_N; j++)
		{
				accu[i][j] = 0.0f;
		}
	}
		
	__syncthreads();

	// load from shared memory to register
	float val_G[BLOCK_SIZE_M][BLOCK_SIZE_K];
	float val_Y[BLOCK_SIZE_K][BLOCK_SIZE_N];
	
	const int loopIdx = 64 / BLOCK_SIZE_K;
	int iTile = 1;
	while (iTile <= loopIdx)
	{
		#pragma unroll
		for (i = 0; i < BLOCK_SIZE_M; i++)
		{
			#pragma unroll
			for (j = 0; j < BLOCK_SIZE_K; j++)
			{
				val_G[i][j] = Gs[yIdx * BLOCK_SIZE_M + i][j];
			}
		}
		
		#pragma unroll
		for (i = 0; i < BLOCK_SIZE_K; i++)
		{
			#pragma unroll
			for (j = 0; j < BLOCK_SIZE_N; j++)
			{
				val_Y[i][j] = Ys[(iTile - 1) * BLOCK_SIZE_K + i][xIdx * BLOCK_SIZE_N + j];		
			}
		}

		// matrix multiply
		#pragma unroll
		for (i = 0; i < BLOCK_SIZE_M; i++)
		{
			#pragma unroll
			for (j = 0; j < BLOCK_SIZE_N; j++)
			{
				#pragma unroll
				for (k = 0; k < BLOCK_SIZE_K; k++)
				{
					accu[i][j] += val_G[i][k] * val_Y[k][j];
				}
			}
		}
		

		
		if (iTile != loopIdx)
		{
			// next G data: BLOCK_SIZE_M * BLOCK_SIZE_K 
			#pragma unroll
			for (i = 0; i < BLOCK_SIZE_M; i++)
			{
				#pragma unroll
				for (j = 0; j < xUnit; j++)
				{
				
					int mIdx = yIdx * BLOCK_SIZE_M + i;
					// note: int kIdx = /*xIdx * BLOCK_SIZE_K +*/ j;
					int kIdx = xIdx * xUnit + j; // BLOCK_SIZE_K: 1, 2, 4
					
					sLoadIdx = 64 * mIdx + BLOCK_SIZE_K * iTile + kIdx;
					
					Gs[mIdx][kIdx] = G0[sLoadIdx];
				}
			}
				
			__syncthreads();
			
		}
		
		iTile++;
	}
	
	int storeIdx;
	#pragma unroll
	for (i = 0; i < BLOCK_SIZE_M; i++)
	{
		#pragma unroll
		for (j = 0; j < BLOCK_SIZE_N; j++)
		{
			storeIdx = (yIdx * BLOCK_SIZE_M + i) * 4
						+ xIdx * BLOCK_SIZE_N + j;
			X0[storeIdx] = accu[i][j];
		}
	}
	
	

	return;
}

template <
    const int BLOCK_SIZE_M,  // height of block of X that each thread calculate, i.e. bm
    const int BLOCK_SIZE_K,  // width of block of G that each thread load into shared memory, i.e. bk
    const int BLOCK_SIZE_N  // width of block of X that each thread calculate i.e. bn
    > 
__global__ void mimo64_block_revised_gpu_kernel(float *G, float *Y, float *X, int nElem)
{
	int i, j, k;
	
	// inside a block
	int xIdx = threadIdx.x; 
	int yIdx = threadIdx.y;
	// block shape
	int xLen = blockDim.x;
	int yLen = blockDim.y;
	// block index
	int Block_X_Idx = blockIdx.x;
	int Block_Y_Idx = blockIdx.y; // 0
	
	int tid_x = Block_X_Idx * xLen + xIdx;
	int tid_y = Block_Y_Idx * yLen + yIdx;

	int xUnit = BLOCK_SIZE_K / xLen; // BLOCK_SIZE_K is multiple of xLen

	// shared memory
	__shared__ float Gs[64][BLOCK_SIZE_K]; // shared memory of G: 256 bytes x BLOCK_SIZE_K
	__shared__ float Ys[64][4]; // shared memory of Y: 1 Kbyte
	
	float accu[BLOCK_SIZE_M][BLOCK_SIZE_N];
	float *G0, *Y0, *X0;

	G0 = G + 64 * 64 * Block_X_Idx;
	Y0 = Y + 64 * 4 * Block_X_Idx;
	X0 = X + 64 * 4 * Block_X_Idx;

	// load data from global memory to shared memeory
	// load the whole Y 
	int sLoadIdx;
	float *Gs_0, *Ys_0;
	Gs_0 = &Gs[0][0];
	Ys_0 = &Ys[0][0];
	
	// printf("nElemPerThread: %d x/y Len: %d %d. currentThreadIdx: %d \n", nElemPerThread, xIdx, yIdx, currentThreadIdx);
	int loadGap = xLen * yLen; // multple of 32s
	int nG_ElemPerThread = (64 * BLOCK_SIZE_K) / loadGap;
	int currentThreadIdx = yIdx * xLen + xIdx;

	int yIdxK = currentThreadIdx / BLOCK_SIZE_K;
	int xIdxK = currentThreadIdx  -  yIdxK * BLOCK_SIZE_K;

	int sGLoadIdx = yIdxK * 64 + xIdxK;
	sLoadIdx = currentThreadIdx;
	int G_ElemGap = 64 / nG_ElemPerThread;
	//yIdxK += G_ElemGap;
	for (i = 0; i < nG_ElemPerThread; i++)
	{
		Gs_0[sLoadIdx] = G0[sGLoadIdx];
		sLoadIdx += loadGap;
		//sGLoadIdx = yIdxK * 64 + xIdxK;
		sGLoadIdx += G_ElemGap * 64;
	}

	//printf("hello world \n");
#if 0
	#pragma unroll
	for (i = 0; i < BLOCK_SIZE_M; i++)
	{
	    #pragma unroll
		for (j = 0; j < xUnit; j++)
		{
			int mIdx = yIdx * BLOCK_SIZE_M + i;
			// note: int kIdx = /*xIdx * BLOCK_SIZE_K +*/ j;
			int kIdx = xIdx * xUnit + j; // BLOCK_SIZE_K: 1, 2, 4
			
			sLoadIdx = 64 * mIdx + kIdx;
			
			Gs[mIdx][kIdx] = G0[sLoadIdx];
		}
	}
#endif

	int nElemPerThread = 64 * 4 / loadGap;

	sLoadIdx = currentThreadIdx;
	for (i = 0; i < nElemPerThread; i++)
	{
		Ys_0[sLoadIdx] = Y0[sLoadIdx];
		sLoadIdx += loadGap;
	}

	// reset the accumulation
	#pragma unroll
	for (i = 0; i < BLOCK_SIZE_M; i++)
	{
		#pragma unroll
		for (j = 0; j < BLOCK_SIZE_N; j++)
		{
				accu[i][j] = 0.0f;
		}
	}
		
	__syncthreads();
	
	// load from shared memory to register
	//float val_G[BLOCK_SIZE_M][BLOCK_SIZE_K];
	//float val_Y[BLOCK_SIZE_K][BLOCK_SIZE_N];
	float val_G[BLOCK_SIZE_M];
	float val_Y[BLOCK_SIZE_N];
	
	const int loopIdx = 64 / BLOCK_SIZE_K;
	int iTile = 1;
	int G_offset = yIdx * BLOCK_SIZE_M;
	int y_offset = xIdx * BLOCK_SIZE_N;
	while (iTile <= loopIdx)
	{
		// matrix multiply	
		#pragma unroll
		for (k = 0; k < BLOCK_SIZE_K; k++)
		{
			#pragma unroll
			for (i = 0; i < BLOCK_SIZE_M; i++)
			{
				val_G[i] = Gs[G_offset + i][k];
			}
			
			#pragma unroll
			for (j = 0; j < BLOCK_SIZE_N; j++)
			{
				val_Y[j] = Ys[(iTile - 1) * BLOCK_SIZE_K + k][y_offset + j];			
			}
			
			
			#pragma unroll
			for (i = 0; i < BLOCK_SIZE_M; i++)
			{
				#pragma unroll
				for (j = 0; j < BLOCK_SIZE_N; j++)
				{
					accu[i][j] += val_G[i] * val_Y[j];

				}
			}
		}
		

		
		if (iTile != loopIdx)
		{
			// next G data: BLOCK_SIZE_M * BLOCK_SIZE_K 
if (1) 
{
			#pragma unroll
			for (i = 0; i < BLOCK_SIZE_M; i++)
			{
				#pragma unroll
				for (j = 0; j < xUnit; j++)
				{
				
					int mIdx = yIdx * BLOCK_SIZE_M + i;
					// note: int kIdx = /*xIdx * BLOCK_SIZE_K +*/ j;
					int kIdx = xIdx * xUnit + j; // BLOCK_SIZE_K: 1, 2, 4
					
					sLoadIdx = 64 * mIdx + BLOCK_SIZE_K * iTile + kIdx;
					
					Gs[mIdx][kIdx] = G0[sLoadIdx];
				}
			}
}

else 
{
			sLoadIdx = yIdx * xLen + xIdx;;


			yIdxK = sLoadIdx / BLOCK_SIZE_K;
			xIdxK = sLoadIdx  -  yIdxK * BLOCK_SIZE_K;
			
			sGLoadIdx = yIdxK * 64 + xIdxK + BLOCK_SIZE_K * iTile;


			//if ((blockIdx.x == 0) && (blockIdx.y == 0))
			//	printf("iTile: %d yIdxK: %d xIdxK: %d \n", iTile, yIdxK, xIdxK);
			
			for (i = 0; i < nG_ElemPerThread; i++)
			{
				Gs_0[sLoadIdx] = G0[sGLoadIdx];
				sLoadIdx += loadGap;
				//sGLoadIdx = yIdxK * 64 + xIdxK;
				sGLoadIdx += G_ElemGap * 64;
			}
}

			__syncthreads();
		}
		
		iTile++;
	}
	
	int storeIdx;
	#pragma unroll
	for (i = 0; i < BLOCK_SIZE_M; i++)
	{
		#pragma unroll
		for (j = 0; j < BLOCK_SIZE_N; j++)
		{
			storeIdx = (yIdx * BLOCK_SIZE_M + i) * 4
						+ xIdx * BLOCK_SIZE_N + j;
			X0[storeIdx] = accu[i][j];
		}
	}
	
	

	return;
}


const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

__global__ void mimo64_wmma_naive16_gpu_kernel(float16_t *G, float16_t *Y, float *X, int nElem)
{	
	wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
	
    // init accumulation
    wmma::fill_fragment(c_frag, 0.0f);
    
	int warpSize = 32;
    // find the warp id
	int tid = threadIdx.y * blockDim.x + threadIdx.x;
	int warpID = tid >> 5;
	float16_t *G0, *Y0;
	float *X0;
	int Block_X_Idx = blockIdx.x;

	G0 = G + 64 * 64 * Block_X_Idx;
	Y0 = Y + 64 * 16 * Block_X_Idx;
	X0 = X + 64 * 16 * Block_X_Idx;

	// printf("tid: %d \n", tid);
	int M = 64;
	int N = 16;
	int K = 64;
    // main loop
    for (int k_step = 0; k_step < K; k_step += WMMA_K) {
        // load data
        wmma::load_matrix_sync(a_frag, G0 + warpID * WMMA_M * K + k_step, K);
        wmma::load_matrix_sync(b_frag, Y0 + k_step * N, N);
        
        // multiplication excution
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
    
    // result store
    wmma::store_matrix_sync(X0 + warpID * WMMA_M * N, c_frag, N, wmma::mem_row_major);
	
	return;
}


__global__ void mimo64_wmma_naive16_col_gpu_kernel(float16_t *G, float16_t *Y, float *X, int nElem)
{	
	wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
	
    // init accumulation
    wmma::fill_fragment(c_frag, 0.0f);
    
	int warpSize = 32;
    // find the warp id
	int tid = threadIdx.y * blockDim.x + threadIdx.x;
	int warpID = tid >> 5;
	float16_t *G0, *Y0;
	float *X0;
	int Block_X_Idx = blockIdx.x;

	G0 = G + 64 * 64 * Block_X_Idx;
	Y0 = Y + 64 * 16 * Block_X_Idx;
	X0 = X + 64 * 16 * Block_X_Idx;

	// printf("tid: %d \n", tid);
	int M = 64;
	int N = 16;
	int K = 64;
    // main loop
    for (int k_step = 0; k_step < K; k_step += WMMA_K) {
        // load data
        wmma::load_matrix_sync(a_frag, Y0 + k_step * N, N);
        wmma::load_matrix_sync(b_frag, G0 + warpID * WMMA_M * K + k_step, K);
        
        // multiplication excution
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
    
    // result store
    wmma::store_matrix_sync(X0 + warpID * WMMA_M * N, c_frag, N, wmma::mem_col_major);
	
	return;
}

__global__ void mimo64_wmma_block16_gpu_kernel(float16_t *G, float16_t *Y, float *X, int nElem)
{
	int i;  
	// shared memory
	__shared__ float16_t Gs_mma[64][16]; // shared memory of G: 2 Kbyte
	__shared__ float16_t Ys_mma[64][16]; // shared memory of Y: 2 Kbyte

	float16_t *G0, *Y0;
	float *X0;
	int Block_X_Idx = blockIdx.x;

	G0 = G + 64 * 64 * Block_X_Idx;
	Y0 = Y + 64 * 16 * Block_X_Idx;
	X0 = X + 64 * 16 * Block_X_Idx;

	int warpSize = 32;
    // find the warp id
    //int warp_row = (blockIdx.y * blockDim.y + threadIdx.y) / warpSize;
    //int warp_col = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
	int warpID = tid >> 5;
	
	int M = 64;
	int N = 16;
	int K = 64;
		
	int blockSize = blockDim.x * blockDim.y; // multple of 32s
	float16_t *Gs_mma_0, *Ys_mma_0;
	Gs_mma_0 = &Gs_mma[0][0];
	Ys_mma_0 = &Ys_mma[0][0];
	
	// load Y0 to shared memory
	int nElemPerThread = 64 * 16 / blockSize;

	int sLoadIdx = tid;
	int YLoadIdx_0 = tid >> 4;
	int YLoadIdx_1 = tid & 15;
	
	for (i = 0; i < nElemPerThread; i++)
	{
		Ys_mma[YLoadIdx_0][YLoadIdx_1] = Y0[sLoadIdx];
	
		sLoadIdx += blockSize;
		YLoadIdx_0 += (blockSize >> 4);
	}

	// load G0 to shared memory
	nElemPerThread = 64 * 16 / blockSize;

	wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

	// init accumulation
    wmma::fill_fragment(c_frag, 0.0f);
  
    // main loop
    for (int k_step = 0; k_step < K; k_step += WMMA_K) {
		sLoadIdx = tid;
		YLoadIdx_0 = tid >> 4;
		YLoadIdx_1 = tid & 15;
		
        for (i = 0; i < nElemPerThread; i++)
		{
			Gs_mma[YLoadIdx_0][YLoadIdx_1] = G0[YLoadIdx_0 * 64 + k_step + YLoadIdx_1];
			//sLoadIdx += blockSize;
			YLoadIdx_0 += (blockSize >> 4);
		}

		__syncthreads();
		
		// load data
        wmma::load_matrix_sync(a_frag, Gs_mma_0 + warpID * WMMA_M * WMMA_K, WMMA_K);
        wmma::load_matrix_sync(b_frag, Ys_mma_0 + k_step * N, N);
        
        // multiplication excution
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
    
    // result store
    wmma::store_matrix_sync(X0 + warpID * WMMA_M * N, c_frag, N, wmma::mem_row_major);
	
	return;
}


int main(int argc, char **argv)
{
    printf("> %s Starting...\n", argv[0]);
	
    // set up data size of vectors
    int nElem = 273 * 12 * 14;
	const int nElemPerMatrix = 16;
    printf("> vector size = %d\n", nElem);

    float *G;
	float *Y;
	float *N0;
	float *X_base;
	float *X;
	float16_t *G_f16;
	float16_t *Y_f16;
	
	
	//G = (float *)malloc((nElem / nElemPerMatrix) * 64 * 64 * sizeof(float));
	//Y = (float *)malloc(nElem * 64 * sizeof(float));
//	N0 = (float *)malloc(nElem * 64 * sizeof(float));
	//X = (float *)malloc(nElem * 64 * sizeof(float));
	//X_base = (float *)malloc(nElem * 64 * sizeof(float));

	//int16_t *h_ScaleLUT;
	Mimo64_alloc_host_mem((void **)&G, (nElem / nElemPerMatrix) * 64 * 64 * sizeof(float));
	Mimo64_alloc_host_mem((void **)&Y, nElem * 64 * sizeof(float));
	Mimo64_alloc_host_mem((void **)&X_base, nElem * 64 * sizeof(float)); 
	Mimo64_alloc_host_mem((void **)&X, nElem * 64 * sizeof(float));
	Mimo64_alloc_host_mem((void **)&G_f16, (nElem / nElemPerMatrix) * 64 * 64 * sizeof(float));
	Mimo64_alloc_host_mem((void **)&Y_f16, nElem * 64 * sizeof(float));


	float *d_G;
	float *d_Y;
	float *d_X;
	
	float16_t *d_G16;
	float16_t *d_Y16;

	Mimo64_alloc_device_mem((void **)&d_G, (nElem / nElemPerMatrix) * 64 * 64 * sizeof(float));
	Mimo64_alloc_device_mem((void **)&d_Y, nElem * 64 * sizeof(float));
	Mimo64_alloc_device_mem((void **)&d_X, nElem * 64 * sizeof(float));
	
	Mimo64_alloc_device_mem((void **)&d_G16, (nElem / nElemPerMatrix) * 64 * 64 * sizeof(float16_t));
	Mimo64_alloc_device_mem((void **)&d_Y16, nElem * 64 * sizeof(float16_t));


    memset(X, 0, nElem * 64 * sizeof(float));
    memset(X_base,  0, nElem * 64 * sizeof(float));

 	initialData_f32(G, (nElem / nElemPerMatrix) * 64 * 64);
	initialData_Y_f32(Y, nElem * 64);

	//mimo64_naive_kernel(G, Y, X_base, nElem, nElemPerMatrix);
	long t_start = useconds();

	mimo64_naive_kernel(G, Y, X_base, nElem, nElemPerMatrix);
	
	long t_end = useconds();
	printf("mimo64_naive_kernel() costs %ld us \n", (t_end - t_start) );

    float kernel_time;
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));


#if 0
    dim3 block (4, 64);
    dim3 grid  ((nElem + block.x - 1) / block.x);
	
	CHECK(hipMemcpy(d_G, G, (nElem / nElemPerMatrix) * 64 * 64 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Y, Y, nElem * 64 * sizeof(float), hipMemcpyHostToDevice));
	

  	CHECK(hipEventRecord(start, 0));

    mimo64_naive_gpu_kernel<<<grid, block>>>(d_G, d_Y, d_X, nElem);

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    CHECK(hipMemcpy(X, d_X, nElem * 64 * sizeof(float), hipMemcpyDeviceToHost));

	printf("mimo64_naive_gpu_kernel() costs %ld us \n", (long)(kernel_time * 1000.0f));

	CHECK(hipMemcpy(d_G, G, (nElem / nElemPerMatrix) * 64 * 64 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Y, Y, nElem * 64 * sizeof(float), hipMemcpyHostToDevice));

  	CHECK(hipEventRecord(start, 0));

    mimo64_naive_gpu_kernel<<<grid, block>>>(d_G, d_Y, d_X, nElem);

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    CHECK(hipMemcpy(X, d_X, nElem * 64 * sizeof(float), hipMemcpyDeviceToHost));

	printf("mimo64_naive_gpu_kernel() costs %ld us \n", (long)(kernel_time * 1000.0f));

	checkResult(X_base, X, nElem * 64);
#endif	

#if 1
	memset(X, 0, nElem * 64 * sizeof(float));
    CHECK(hipMemcpy(d_X, X, nElem * 64 * sizeof(float), hipMemcpyHostToDevice));

	memset(Y_f16, 0, nElem * 64 * sizeof(float));

	float2half(G_f16, G, (nElem / nElemPerMatrix) * 64 * 64);
	float2half(Y_f16, Y, nElem * 64);

	CHECK(hipMemcpy(d_G16, G_f16, (nElem / nElemPerMatrix) * 64 * 64 * sizeof(float16_t), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Y16, Y_f16, nElem * 64 * sizeof(float16_t), hipMemcpyHostToDevice));

  	CHECK(hipEventRecord(start, 0));
	
	dim3 block_wmma (128);
    dim3 grid_wmma  ((nElem + 15) / 16); // 4 elements in one block

  	CHECK(hipEventRecord(start, 0));

    mimo64_wmma_naive16_gpu_kernel<<<grid_wmma, block_wmma>>>(d_G16, d_Y16, d_X, nElem);

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    CHECK(hipMemcpy(X, d_X, nElem * 64 * sizeof(float), hipMemcpyDeviceToHost));

	printf("mimo64_wmma_naive16_gpu_kernel() costs %ld us \n", (long)(kernel_time * 1000.0f));

	checkResult(X_base, X, nElem * 64);	
#endif

#if 1
	memset(X, 0, nElem * 64 * sizeof(float));
    CHECK(hipMemcpy(d_X, X, nElem * 64 * sizeof(float), hipMemcpyHostToDevice));

	memset(Y_f16, 0, nElem * 64 * sizeof(float));

	float2half(G_f16, G, (nElem / nElemPerMatrix) * 64 * 64);
	float2half(Y_f16, Y, nElem * 64);

	CHECK(hipMemcpy(d_G16, G_f16, (nElem / nElemPerMatrix) * 64 * 64 * sizeof(float16_t), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Y16, Y_f16, nElem * 64 * sizeof(float16_t), hipMemcpyHostToDevice));

  	CHECK(hipEventRecord(start, 0));
	
	// dim3 block_wmma (128);
    // dim3 grid_wmma  ((nElem + 15) / 16); // 4 elements in one block

  	CHECK(hipEventRecord(start, 0));

    mimo64_wmma_naive16_gpu_kernel<<<grid_wmma, block_wmma>>>(d_G16, d_Y16, d_X, nElem);

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    CHECK(hipMemcpy(X, d_X, nElem * 64 * sizeof(float), hipMemcpyDeviceToHost));

	printf("mimo64_wmma_naive16_gpu_kernel() costs %ld us \n", (long)(kernel_time * 1000.0f));

	checkResult(X_base, X, nElem * 64);

	
#endif

#if 1
	memset(X, 0, nElem * 64 * sizeof(float));
    CHECK(hipMemcpy(d_X, X, nElem * 64 * sizeof(float), hipMemcpyHostToDevice));
	
	memset(Y_f16, 0, nElem * 64 * sizeof(float));

	float2half(G_f16, G, (nElem / nElemPerMatrix) * 64 * 64);
	float2half(Y_f16, Y, nElem * 64);

	CHECK(hipMemcpy(d_G16, G_f16, (nElem / nElemPerMatrix) * 64 * 64 * sizeof(float16_t), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Y16, Y_f16, nElem * 64 * sizeof(float16_t), hipMemcpyHostToDevice));

  	CHECK(hipEventRecord(start, 0));
	
	dim3 block_wmma0 (128);
    dim3 grid_wmma0  ((nElem + 15) / 16); // 4 elements in one block

  	CHECK(hipEventRecord(start, 0));

    mimo64_wmma_naive16_col_gpu_kernel<<<grid_wmma0, block_wmma0>>>(d_G16, d_Y16, d_X, nElem);

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    CHECK(hipMemcpy(X, d_X, nElem * 64 * sizeof(float), hipMemcpyDeviceToHost));

	printf("mimo64_wmma_naive16_col_gpu_kernel() costs %ld us \n", (long)(kernel_time * 1000.0f));

	checkResult(X_base, X, nElem * 64);
#endif


#if 1
	memset(X, 0, nElem * 64 * sizeof(float));
    CHECK(hipMemcpy(d_X, X, nElem * 64 * sizeof(float), hipMemcpyHostToDevice));
	
	memset(Y_f16, 0, nElem * 64 * sizeof(float));

	float2half(G_f16, G, (nElem / nElemPerMatrix) * 64 * 64);
	float2half(Y_f16, Y, nElem * 64);

	CHECK(hipMemcpy(d_G16, G_f16, (nElem / nElemPerMatrix) * 64 * 64 * sizeof(float16_t), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Y16, Y_f16, nElem * 64 * sizeof(float16_t), hipMemcpyHostToDevice));

  	CHECK(hipEventRecord(start, 0));
	
	dim3 block_wmma_1 (128);
    dim3 grid_wmma_1  ((nElem + 15) / 16); // 4 elements in one block

  	CHECK(hipEventRecord(start, 0));

    mimo64_wmma_block16_gpu_kernel<<<grid_wmma_1, block_wmma_1>>>(d_G16, d_Y16, d_X, nElem);

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    CHECK(hipMemcpy(X, d_X, nElem * 64 * sizeof(float), hipMemcpyDeviceToHost));

	printf("mimo64_wmma_block16_gpu_kernel() costs %ld us \n", (long)(kernel_time * 1000.0f));

	checkResult(X_base, X, nElem * 64);

	
#endif

#if 0
	// memset d_X, X
    memset(X, 0, nElem * 64 * sizeof(float));
    CHECK(hipMemcpy(d_X, X, nElem * 64 * sizeof(float), hipMemcpyHostToDevice));
	
	CHECK(hipMemcpy(d_G, G, (nElem / nElemPerMatrix) * 64 * 64 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Y, Y, nElem * 64 * sizeof(float), hipMemcpyHostToDevice));

	// (M, N, K) = (1, 1, 4), (1, 1, 2), (2, 1, 4), (2, 2, 4), (4, 1, 2), (4, 1, 4), (1, 2, 4), (2, 1, 8)
	const int BLOCK_SIZE_M = 2;
	const int BLOCK_SIZE_N = 1;
	const int BLOCK_SIZE_K = 4;
	
	dim3 block1 (4 / BLOCK_SIZE_N, 64 / BLOCK_SIZE_M);
    dim3 grid1  ((nElem + 3) / 4); // 4 elements in one block

  	CHECK(hipEventRecord(start, 0));

    mimo64_block_naive_gpu_kernel<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N><<<grid1, block1>>>(d_G, d_Y, d_X, nElem);

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    CHECK(hipMemcpy(X, d_X, nElem * 64 * sizeof(float), hipMemcpyDeviceToHost));

	printf("mimo64_block_naive_gpu_kernel() costs %ld us \n", (long)(kernel_time * 1000.0f));

	checkResult(X_base, X, nElem * 64);
#endif

#if 0
	// memset d_X, X
    memset(X, 0, nElem * 64 * sizeof(float));
    CHECK(hipMemcpy(d_X, X, nElem * 64 * sizeof(float), hipMemcpyHostToDevice));

	CHECK(hipMemcpy(d_G, G, (nElem / nElemPerMatrix) * 64 * 64 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Y, Y, nElem * 64 * sizeof(float), hipMemcpyHostToDevice));



	// (M, N, K) = (1, 1, 4), (1, 4, 4), (1, 1, 2), (2, 1, 4), (2, 2, 4), (4, 1, 2), (4, 1, 4), (1, 2, 4), (2, 1, 8) 
	const int BLOCK_SIZE_M_2 = 2;
	const int BLOCK_SIZE_N_2 = 1;
	const int BLOCK_SIZE_K_2 = 4;
	
	dim3 block2 (4 / BLOCK_SIZE_N_2, 64 / BLOCK_SIZE_M_2);
    dim3 grid2  ((nElem + 3) / 4); // 4 elements in one block

	printf("gridIdx.x: %d blockIdx.x: %d blockIdx.y: %d \n", grid2.x, block2.x, block2.y);

  	CHECK(hipEventRecord(start, 0));

    mimo64_block_revised_gpu_kernel<BLOCK_SIZE_M_2, BLOCK_SIZE_K_2, BLOCK_SIZE_N_2><<<grid2, block2>>>(d_G, d_Y, d_X, nElem);

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&kernel_time, start, stop));

    CHECK(hipMemcpy(X, d_X, nElem * 64 * sizeof(float), hipMemcpyDeviceToHost));

	printf("mimo64_block_revised_gpu_kernel() costs %ld us \n", (long)(kernel_time * 1000.0f));

	checkResult(X_base, X, nElem * 64);
#endif


	Mimo64_free_host_mem(G);
	Mimo64_free_host_mem(Y);
	Mimo64_free_host_mem(X_base);
	Mimo64_free_host_mem(X);

	Mimo64_free_device_mem(d_G);
	Mimo64_free_device_mem(d_Y);
	Mimo64_free_device_mem(d_X);
	


	
	return 0;
}